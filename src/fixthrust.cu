#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/extrema.h>
#include <thrust/transform.h>
#include <iostream>

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

template <typename T, typename Op>
struct Operation {

	uint shift_val;

	Operation(uint shift_val) {
		this->shift_val = shift_val;
	}

	__host__ __device__
	T operator()(const T x, const T y)
	{
		T fix = y << shift_val;
		Op op = Op();
		return op(x, fix);
	}
};

void print(thrust::host_vector<uint> h_vec) {
	std::cout << "\n";
	for (uint i = 0; i < h_vec.size(); i++) {
		std::cout << h_vec[i] << " ";
	}
	std::cout << "\n";
}

int main(void) {
	uint num_of_segments;
	uint num_of_elements;

	scanf("%d", &num_of_segments);
	thrust::host_vector<uint> h_seg_aux(num_of_segments + 1);
	for (uint i = 0; i < num_of_segments + 1; i++)
		scanf("%d", &h_seg_aux[i]);

	scanf("%d", &num_of_elements);
	thrust::host_vector<uint> h_vec(num_of_elements);
	for (uint i = 0; i < num_of_elements; i++)
		scanf("%d", &h_vec[i]);

	thrust::host_vector<uint> h_seg(num_of_elements);
	for (uint i = 0; i < num_of_segments; i++) {
		for (uint j = h_seg_aux[i]; j < h_seg_aux[i + 1]; j++) {
			h_seg[j] = i;
		}
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	thrust::device_vector<uint> d_vec(num_of_elements);
	thrust::device_vector<uint> d_seg = h_seg;

	for (uint i = 0; i < EXECUTIONS; i++) {
		thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());
		/*
		 * maximum element of the array.
		 */
		hipEventRecord(start);
		thrust::device_vector<uint>::iterator iter = thrust::max_element(d_vec.begin(), d_vec.end());
		uint max_val = *iter;
		uint mostSignificantBit = (uint)log2((double)max_val) + 1;
		/*
		 * add prefix to the elements
		 */
		Operation< uint, thrust::plus<uint> > op_plus(mostSignificantBit);
		thrust::transform(d_vec.begin(), d_vec.end(), d_seg.begin(), d_vec.begin(), op_plus);
		/*
		 * sort the segments
		 */
		thrust::sort(d_vec.begin(), d_vec.end());
		/*
		 * update back the array elements
		 */

		Operation< uint, thrust::minus<uint> > op_minus(mostSignificantBit);
		thrust::transform(d_vec.begin(), d_vec.end(), d_seg.begin(), d_vec.begin(), op_minus);
		hipEventRecord(stop);
		hipEventSynchronize(stop);

		if (ELAPSED_TIME == 1) {
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << milliseconds << "\n";
		}

		hipError_t errSync = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess)
			printf("4: Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("4: Async kernel error: %s\n", hipGetErrorString(errAsync));
	}

	thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

	if (ELAPSED_TIME != 1) {
		print(h_vec);
	}

	return 0;
}
