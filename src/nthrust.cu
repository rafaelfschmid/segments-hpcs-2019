/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

#include <chrono>
#include <iostream>

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

void print(thrust::host_vector<int> h_vec) {
	std::cout << "\n";
	for (int i = 0; i < h_vec.size(); i++) {
		std::cout << h_vec[i] << " ";
	}
	std::cout << "\n";
}

int main(void) {
	int num_of_segments;
	int num_of_elements;
	int i;

	scanf("%d", &num_of_segments);
	thrust::host_vector<int> h_seg(num_of_segments + 1);
	for (i = 0; i < num_of_segments + 1; i++)
		scanf("%d", &h_seg[i]);

	scanf("%d", &num_of_elements);
	thrust::host_vector<int> h_vec(num_of_elements);
	for (i = 0; i < num_of_elements; i++)
		scanf("%d", &h_vec[i]);

	thrust::device_vector<uint> d_vec(num_of_elements);

	for (uint i = 0; i < EXECUTIONS; i++) {
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());

		hipEventRecord(start);
		for (int i = 0; i < num_of_segments; i++) {
			thrust::sort(d_vec.begin() + h_seg[i],
					d_vec.begin() + h_seg[i + 1]);
		}
		hipEventRecord(stop);

		if (ELAPSED_TIME == 1) {
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << milliseconds << "\n";
		}

		hipDeviceSynchronize();
	}

	thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

	if (ELAPSED_TIME != 1) {
		print(h_vec);
	}

	return 0;
}
