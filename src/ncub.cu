/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_reduce.cuh>

#include <iostream>

typedef unsigned int uint;

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 512
#endif

#ifndef NUM_STREAMS
#define NUM_STREAMS 4
#endif

void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit (EXIT_FAILURE);
	}

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("1: Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("1: Async kernel error: %s\n", hipGetErrorString(errAsync));
}

void print(uint* host_data, uint n) {
	std::cout << "\n";
	for (uint i = 0; i < n; i++) {
		std::cout << host_data[i] << " ";
	}
	std::cout << "\n";
}

int main(void) {
	uint num_of_segments;
	uint num_of_elements;
	uint i;

	scanf("%d", &num_of_segments);
	uint mem_size_seg = sizeof(uint) * (num_of_segments + 1);
	uint *h_seg = (uint *) malloc(mem_size_seg);
	for (i = 0; i < num_of_segments + 1; i++)
		scanf("%d", &h_seg[i]);

	scanf("%d", &num_of_elements);
	int mem_size_vec = sizeof(uint) * num_of_elements;
	uint *h_vec = (uint *) malloc(mem_size_vec);
	uint *h_value = (uint *) malloc(mem_size_vec);
	for (i = 0; i < num_of_elements; i++) {
		scanf("%d", &h_vec[i]);
		h_value[i] = i;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	uint *d_vec, *d_vec_out;

	cudaTest(hipMalloc((void **) &d_vec, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_vec_out, mem_size_vec));

	void *d_temp1 = NULL;
	size_t temp_bytes1 = 0;

	void *d_temp2 = NULL;
	size_t temp_bytes2 = 0;

	void *d_temp3 = NULL;
	size_t temp_bytes3 = 0;

	void *d_temp4 = NULL;
	size_t temp_bytes4 = 0;

	int num_of_streams = NUM_STREAMS;

	if(num_of_streams > num_of_segments){
		num_of_streams = num_of_segments;
	}

	hipStream_t streams[NUM_STREAMS];
	for(int i = 0; i < NUM_STREAMS; i++) {
		hipStreamCreate(&streams[i]);
	}

	for (uint e = 0; e < EXECUTIONS; e++) {
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		cudaTest(hipMemcpy(d_vec, h_vec, mem_size_vec, hipMemcpyHostToDevice));

		hipEventRecord(start);
		for (int i = 0; i < num_of_segments; i+=num_of_streams) {
			//for (int s = 0; s < num_of_streams; s++) {
			int s=0;
			hipcub::DeviceRadixSort::SortKeys(d_temp1, temp_bytes1, d_vec+h_seg[i+s], d_vec_out+h_seg[i+s],
					h_seg[i+1+s]-h_seg[i+s], 0, sizeof(uint)*8, streams[s]);
			hipMalloc((void **) &d_temp1, temp_bytes1);
			hipcub::DeviceRadixSort::SortKeys(d_temp1, temp_bytes1, d_vec+h_seg[i+s], d_vec_out+h_seg[i+s],
					h_seg[i+1+s]-h_seg[i+s], 0, sizeof(uint)*8, streams[s]);

			s=1;
			hipcub::DeviceRadixSort::SortKeys(d_temp2, temp_bytes2, d_vec+h_seg[i+s], d_vec_out+h_seg[i+s],
									h_seg[i+1+s]-h_seg[i+s], 0, sizeof(uint)*8, streams[s]);
			hipMalloc((void **) &d_temp2, temp_bytes2);
			hipcub::DeviceRadixSort::SortKeys(d_temp2, temp_bytes2, d_vec+h_seg[i+s], d_vec_out+h_seg[i+s],
					h_seg[i+1+s]-h_seg[i+s], 0, sizeof(uint)*8, streams[s]);

			s=2;
			hipcub::DeviceRadixSort::SortKeys(d_temp3, temp_bytes3, d_vec+h_seg[i+s], d_vec_out+h_seg[i+s],
									h_seg[i+1+s]-h_seg[i+s], 0, sizeof(uint)*8, streams[s]);
			hipMalloc((void **) &d_temp3, temp_bytes3);
			hipcub::DeviceRadixSort::SortKeys(d_temp3, temp_bytes3, d_vec+h_seg[i+s], d_vec_out+h_seg[i+s],
					h_seg[i+1+s]-h_seg[i+s], 0, sizeof(uint)*8, streams[s]);

			s=3;
			hipcub::DeviceRadixSort::SortKeys(d_temp4, temp_bytes4, d_vec+h_seg[i+s], d_vec_out+h_seg[i+s],
									h_seg[i+1+s]-h_seg[i+s], 0, sizeof(uint)*8, streams[s]);
			hipMalloc((void **) &d_temp4, temp_bytes4);
			hipcub::DeviceRadixSort::SortKeys(d_temp4, temp_bytes4, d_vec+h_seg[i+s], d_vec_out+h_seg[i+s],
					h_seg[i+1+s]-h_seg[i+s], 0, sizeof(uint)*8, streams[s]);
		//}
		}
		hipEventRecord(stop);
		hipEventSynchronize(stop);

		hipError_t errSync = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess)
			printf("4: Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("4: Async kernel error: %s\n", hipGetErrorString(errAsync));

		if (ELAPSED_TIME == 1) {
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << milliseconds << "\n";
		}

		hipDeviceSynchronize();

		hipFree(d_temp1);
		temp_bytes1 = 0;
		d_temp1 = NULL;

		hipFree(d_temp2);
		temp_bytes2 = 0;
		d_temp2 = NULL;

		hipFree(d_temp3);
		temp_bytes3 = 0;
		d_temp3 = NULL;

		hipFree(d_temp4);
		temp_bytes4 = 0;
		d_temp4 = NULL;
	}

	hipMemcpy(h_vec, d_vec_out, mem_size_vec, hipMemcpyDeviceToHost);

	hipFree(streams);
	hipFree(d_vec);
	hipFree(d_vec_out);

	if (ELAPSED_TIME != 1) {
		print(h_vec, num_of_elements);
	}

	free(h_seg);
	free(h_vec);
	free(h_value);

	return 0;
}
