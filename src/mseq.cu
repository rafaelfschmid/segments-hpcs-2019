#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================

 COMPILAR USANDO O SEGUINTE COMANDO:

 nvcc segmented_sort.cu -o segmented_sort -std=c++11 --expt-extended-lambda -I"/home/schmid/Dropbox/Unicamp/workspace/sorting_segments/moderngpu-master/src"

 */

#include <stdio.h>
#include <stdlib.h>
#include <algorithm>    // std::sort
#include <hip/hip_runtime.h>
#include <iostream>


#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

void print(uint* host_data, uint n) {
	std::cout << "\n";
	for (uint i = 0; i < n; i++) {
		std::cout << host_data[i] << " ";
	}
	std::cout << "\n";
}

void printSeg(uint* host_data, uint num_seg, uint num_ele) {
	std::cout << "\n";
	for (uint i = 0; i < num_seg; i++) {
		std::cout << host_data[i] << " ";
	}
	std::cout << num_ele << " ";
	std::cout << "\n";
}

void segmented_sorting(uint* vec, uint* seg, int number_of_segments) {

	for(int i = 0; i < number_of_segments; i++) {
		std::stable_sort (&vec[seg[i]], &vec[seg[i+1]]);
	}
}

int main(int argc, char** argv) {

	uint num_of_segments;
	uint num_of_elements;
	uint i;

	scanf("%d", &num_of_segments);
	uint mem_size_seg = sizeof(uint) * (num_of_segments + 1);
	uint *h_seg = (uint *) malloc(mem_size_seg);
	for (i = 0; i < num_of_segments + 1; i++)
		scanf("%d", &h_seg[i]);

	scanf("%d", &num_of_elements);
	uint mem_size_vec = sizeof(uint) * num_of_elements;
	uint *h_vec_aux = (uint *) malloc(mem_size_vec);
	for (i = 0; i < num_of_elements; i++)
		scanf("%d", &h_vec_aux[i]);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	uint *h_vec = (uint *) malloc(mem_size_vec);

	for (uint j = 0; j < EXECUTIONS; j++) {

		for (i = 0; i < num_of_elements; i++)
			h_vec[i] = h_vec_aux[i];

		hipEventRecord(start);
		segmented_sorting(h_vec, h_seg, num_of_segments);
		hipEventRecord(stop);

		if (ELAPSED_TIME == 1) {
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << milliseconds << "\n";
		}

	}

	if (ELAPSED_TIME != 1) {
		print(h_vec, num_of_elements);
	}

	free(h_seg);
	free(h_vec);
	free(h_vec_aux);

	return 0;
}

/***
 * SEGMENTED SORT FUNCIONANDO
 *
 *
 uint n = atoi(argv[1]);
 uint m = atoi(argv[2]);
 uint num_segments = n / m;
 mgpu::standard_context_t context;
 rand_key<uint> func(m);

 mgpu::mem_t<uint> segs = mgpu::fill_function(func, num_segments, context);
 //mgpu::mem_t<uint> segs = mgpu::fill_random(0, n - 1, num_segments, true, context);
 std::vector<uint> segs_host = mgpu::from_mem(segs);
 mgpu::mem_t<uint> data = mgpu::fill_random(0, pow(2, NUMBER_BITS_SIZE), n,
 false, context);
 mgpu::mem_t<uint> values(n, context);
 std::vector<uint> data_host = mgpu::from_mem(data);

 //	print(segs_host); print(data_host);

 mgpu::segmented_sort(data.data(), values.data(), n, segs.data(),
 num_segments, mgpu::less_t<uint>(), context);

 std::vector<uint> sorted = from_mem(data);
 std::vector<uint> indices_host = from_mem(values);

 std::cout << "\n";
 //print(segs_host);
 //	print(data_host); print(indices_host);
 *
 */
